#include "hip/hip_runtime.h"
#define EIGEN_USE_GPU
#include "third_party/eigen3/unsupported/Eigen/CXX11/Tensor"
#include "transform.h"
#include<hip/hip_runtime_api.h>
#include "math.h"
#include <hip/hip_runtime.h>

using namespace std;

#define stepSize 0.125f
#define threadsPerBlockRayTrace 256

texture<float, 2, hipReadModeElementType> imgTexture2D;

static int _nViews, _nCols, _nX, _nY, _nImageLength, _nProjectionLength, _nSample;
static float _D, _R, _dRange, _dAngle, _rFOV, _dx, _dy, _xmin, _ymin, _dCol, _dLeft, _nViewSize, _nImageSize, _nProjectionSize;
__constant__ int _nViews_, _nCols_, _nX_, _nY_, _nSample_;
__constant__ float _D_, _R_, _dRange_, _dAngle_, _rFOV_, _dx_, _dy_, _xmin_, _ymin_, _dCol_, _dLeft_;

__host__ void
para_setup_fan_curve(float D, float R, int nViews, float dRange, float dAngle, int nCols, int nX, int nY, float rFOV, float dx, float dy, float xmin, float ymin, float dCol, float dLeft, int nSamp)
{

   _D = D;
   _R = R;
   _nViews = nViews;
   _dRange = dRange;
   _dAngle = dAngle;
   _nCols = nCols;
   _nX = nX;
   _nY = nY;
   _rFOV = rFOV;
   _dx = dx;
   _dy = dy;
   _xmin = xmin;
   _ymin = ymin;
   _dCol = dCol;
   _dLeft = dLeft;
   _nViewSize = nCols*sizeof(float);
   _nImageLength = nX*nY;
   _nImageSize = nX*nY*sizeof(float);
   _nProjectionLength = nCols*nViews;
   _nProjectionSize = nCols*nViews*sizeof(float);
   _nSample = nSamp;

   checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(_D_), &D, sizeof(float) ) );
   checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(_R_), &R, sizeof(float) ) );
   checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(_nViews_), &nViews, sizeof(int) ) );
   checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(_dRange_), &dRange, sizeof(float) ) );
   checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(_dAngle_), &dAngle, sizeof(float) ) );
   checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(_nCols_), &nCols, sizeof(int) ) );
   checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(_nX_), &nX, sizeof(int) ) );
   checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(_nY_), &nY, sizeof(int) ) );
   checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(_rFOV_), &rFOV, sizeof(float) ) );
   checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(_dx_), &dx, sizeof(float) ) );
   checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(_dy_), &dy, sizeof(float) ) );
   checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(_xmin_), &xmin, sizeof(float) ) );
   checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(_ymin_), &ymin, sizeof(float) ) );
   checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(_dCol_), &dCol, sizeof(float) ) );
   checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(_dLeft_), &dLeft, sizeof(float) ) );
   checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(_nSample_), &nSamp, sizeof(int) ) );
}

__global__ void
GenerateRayTraceMap_FB_ReconImage_OneView_kernel(float fSourceToCenterDistance, float fAngle, float fColMin, float fDeltaCol, float fFOVRadius, float * pfStart1, float * pfStart2, float * pfStep1, float * pfStep2, int * pnSteps, float * pfDistIntersection)
{
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    float fRayAngle = fAngle+fColMin+(i+0.5f)*fDeltaCol;
    float fSinAngle = sinf(fRayAngle);
    float fCosAngle = cosf(fRayAngle);

    float fDistanceToCenterRay = fSourceToCenterDistance*sinf(fColMin+(i+0.5f)*fDeltaCol);

    float intersection1[2],intersection2[2];

    float fDistIntersection = 0;
    int numSamples;


    float x1,x2,y1,y2;
    x1 = -sqrtf(fFOVRadius*fFOVRadius-fDistanceToCenterRay*fDistanceToCenterRay);
    x2 = -x1;
    y1 = -fDistanceToCenterRay;
    y2 = -fDistanceToCenterRay;

    intersection1[0] = x1*fCosAngle-y1*fSinAngle;
    intersection1[1] = x1*fSinAngle+y1*fCosAngle;
    intersection2[0] = x2*fCosAngle-y2*fSinAngle;
    intersection2[1] = x2*fSinAngle+y2*fCosAngle;

    fDistIntersection = (float)sqrt((intersection1[0]-intersection2[0])*(intersection1[0]-intersection2[0])+(intersection1[1]-intersection2[1])*(intersection1[1]-intersection2[1]));
    numSamples	= int(floor(fDistIntersection/stepSize));

    pnSteps[i] = numSamples;
    pfStep1[i] = (intersection2[0]-intersection1[0])/numSamples;
    pfStep2[i] = (intersection2[1]-intersection1[1])/numSamples;
    pfStart1[i] = intersection1[0]+fFOVRadius;
    pfStart2[i] = intersection1[1]+fFOVRadius;
//	pfDistIntersection[i] = (fDistIntersection<1 ? 1: fDistIntersection);

}

__global__ void
RayTrace_ReconImage_OneView_kernel(float * pfPrj, const float * pfStart1, const float * pfStart2, const float * pfStep1, const float * pfStep2, const int * pnSteps)
{
    unsigned int ni=threadIdx.x;
    unsigned int nCacheIndex=ni;
    unsigned int nCol=blockIdx.x;

    __shared__	float pfPrjCache[threadsPerBlockRayTrace];
    pfPrjCache[nCacheIndex] = 0;

    __shared__ int nLength;
    __shared__ float fStart1;
    __shared__ float fStart2;
    __shared__ float fStep1;
    __shared__ float fStep2;

    nLength=pnSteps[nCol];
    fStart1=pfStart1[nCol];
    fStart2=pfStart2[nCol];
    fStep1=pfStep1[nCol];
    fStep2=pfStep2[nCol];

    while(ni<nLength)
    {
        pfPrjCache[nCacheIndex] += tex2D(imgTexture2D, fStart1+ni*fStep1, fStart2+ni*fStep2);
        ni+=threadsPerBlockRayTrace;
    }
    __syncthreads();
    int i = threadsPerBlockRayTrace/2;
    while ( i!= 0)
    {
        if (nCacheIndex < i)
        {
            pfPrjCache[nCacheIndex] += pfPrjCache[nCacheIndex+i];
        }
        __syncthreads();
        i /= 2;
    }

    if(nCacheIndex == 0)
    {
        pfPrj[nCol] = pfPrjCache[0]*stepSize*_dx_;
    }
}

__host__ void
project_fan_curve_gpu(const float * dev_u, float * dev_v, const float * d_pfStart_1, const float * d_pfStart_2, const float * d_pfStep_1, const float * d_pfStep_2, const int * d_pnSteps, hipArray* cu_ReconImage)
{
    dim3 dimThreadsPerBlockSino(32, 1, 1);
    dim3 dimNumBlocksSino(ceil(_nCols/dimThreadsPerBlockSino.x)+1, 1, 1);
    dim3 dimThreadsPerBlockImg(32, 32, 1);
    dim3 dimNumBlocksImg(_nX / dimThreadsPerBlockImg.x, _nY / dimThreadsPerBlockImg.y, 1);

    float * dev_ybuffer = NULL;
    checkCudaErrors(hipMalloc((void**) &dev_ybuffer, _nViewSize));
    checkCudaErrors(hipMemset(dev_ybuffer, 0, _nViewSize));
    checkCudaErrors(hipMemcpyToArray(cu_ReconImage, 0, 0, dev_u, _nImageSize, hipMemcpyDeviceToDevice));

    int nV, nOffset;
    for(nV=0; nV<_nViews; nV++)
    {
        nOffset = nV*_nCols;
        RayTrace_ReconImage_OneView_kernel<<< _nCols, threadsPerBlockRayTrace>>>
                (dev_ybuffer, d_pfStart_1+nOffset, d_pfStart_2+nOffset, d_pfStep_1+nOffset, d_pfStep_2+nOffset, d_pnSteps+nOffset);
        checkCudaErrors( hipDeviceSynchronize() );
        checkCudaErrors( hipMemcpy( dev_v + nOffset, dev_ybuffer, _nViewSize, hipMemcpyDeviceToDevice ) );
    }
    checkCudaErrors(hipFree(dev_ybuffer));
}

__host__ void
project_fan_curve(const float * image, float * projection, const float * anglePos)
{

    ///////////////////////////////////////////////////////////////////////////////////////////////// CUDA Paralleling parameters //////////////////////////////////////////////////////
    dim3 dimThreadsPerBlockSino(32, 1, 1);
    dim3 dimNumBlocksSino(ceil(_nCols/dimThreadsPerBlockSino.x)+1, 1, 1);
    dim3 dimThreadsPerBlockImg(32, 32, 1);
    dim3 dimNumBlocksImg(_nX / dimThreadsPerBlockImg.x, _nY / dimThreadsPerBlockImg.y, 1);
    dim3 dimThreadsPerBlock(32, 32, 1);                                                                                // Define 2D blocks and grids for matrix computation
    dim3 dimBlocksPerGrid(_nX / dimThreadsPerBlock.x, _nY / dimThreadsPerBlock.y, 1);

    ///////////////////////////////////////////////////////////////////////////////////////////// Initialization in Graphic card //////////////////////////////////////////////////////////////////////////
    float * dev_y = NULL;
    float * d_pfDistIntersections = NULL;
    float * d_pfStart_1 = NULL;
    float * d_pfStart_2 = NULL;
    float * d_pfStep_1 = NULL;
    float * d_pfStep_2 = NULL;
    int * d_pnSteps = NULL;
    float * dev_x = NULL;
    float * dev_u = NULL;
    float * dev_v = NULL;

    //////////////////////////////////////////////////////////////////////////////////////////////////////// Allocate Memory in the Graphic card ///////////////////////////////////////////////////////////////////////////
    checkCudaErrors(hipMalloc((void**) &dev_y, _nProjectionSize*_nSample));
    checkCudaErrors(hipMemset(dev_y, 0, _nProjectionSize*_nSample));
    checkCudaErrors(hipMalloc((void**) &d_pfDistIntersections, _nProjectionSize));
    checkCudaErrors(hipMemset(d_pfDistIntersections, 0, _nProjectionSize));
    checkCudaErrors(hipMalloc((void**) &d_pfStart_1, _nProjectionSize));
    checkCudaErrors(hipMemset(d_pfStart_1, 0, _nProjectionSize));
    checkCudaErrors(hipMalloc((void**) &d_pfStart_2, _nProjectionSize));
    checkCudaErrors(hipMemset(d_pfStart_2, 0, _nProjectionSize));
    checkCudaErrors(hipMalloc((void**) &d_pfStep_1, _nProjectionSize));
    checkCudaErrors(hipMemset(d_pfStep_1, 0, _nProjectionSize));
    checkCudaErrors(hipMalloc((void**) &d_pfStep_2, _nProjectionSize));
    checkCudaErrors(hipMemset(d_pfStep_2, 0, _nProjectionSize));
    checkCudaErrors(hipMalloc((void**) &d_pnSteps, _nProjectionSize));
    checkCudaErrors(hipMemset(d_pnSteps, 0, _nProjectionSize));
    checkCudaErrors(hipMalloc((void**) &dev_x, _nImageSize*_nSample));
    checkCudaErrors(hipMemset(dev_x, 0, _nImageSize*_nSample));
    checkCudaErrors(hipMalloc((void**) &dev_u, _nImageSize));
    checkCudaErrors(hipMemset(dev_u, 0, _nImageSize));
    checkCudaErrors(hipMalloc((void**) &dev_v, _nProjectionSize));
    checkCudaErrors(hipMemset(dev_v, 0, _nProjectionSize));

    ///////////////////////////////////////////////////////////////////////////////////////////////// CUDA Paralleling Texture Mapping Setting  //////////////////////////////////////////////////////
    hipArray * cu_ReconImage = NULL;
    hipChannelFormatDesc channelDescFloat = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    checkCudaErrors(hipMallocArray(&cu_ReconImage, &channelDescFloat, _nX, _nY));

    imgTexture2D.addressMode[0] = hipAddressModeClamp;
    imgTexture2D.addressMode[1] = hipAddressModeClamp;
    imgTexture2D.normalized = false;
    imgTexture2D.filterMode = hipFilterModeLinear;
    checkCudaErrors(hipBindTextureToArray(imgTexture2D, cu_ReconImage, channelDescFloat));

    ////////////////////////////////////////////////////////////////////////////////////// Copy data from host to device ////////////////////////////////////////////////////////////////////////////////////////
    checkCudaErrors(hipMemcpy(dev_x, image, _nImageSize*_nSample, hipMemcpyHostToDevice));

    ///////////////////////////////////////////////////////////////////////////// Perform ForwardProjection ////////////////////////////////////////////////////////////////////
    int nV, nOffset, nS;
    for(nV=0; nV<_nViews; nV++)
    {
        nOffset = nV*_nCols;
        GenerateRayTraceMap_FB_ReconImage_OneView_kernel<<<dimNumBlocksSino, dimThreadsPerBlockSino>>>
            (_R/_dx, anglePos[nV], _dLeft, _dCol, _rFOV/_dx,
            d_pfStart_1+nOffset, d_pfStart_2+nOffset, d_pfStep_1+nOffset,
            d_pfStep_2+nOffset, d_pnSteps+nOffset, d_pfDistIntersections+nOffset);
        checkCudaErrors( hipDeviceSynchronize() );
    }

    for(nS=0; nS<_nSample; nS++)
    {
        checkCudaErrors( hipMemcpy(dev_u, dev_x+_nImageLength*nS, _nImageSize, hipMemcpyDeviceToDevice ) );
        checkCudaErrors(hipMemset(dev_v, 0, _nProjectionSize));
        project_fan_curve_gpu(dev_u, dev_v, d_pfStart_1, d_pfStart_2, d_pfStep_1, d_pfStep_2, d_pnSteps, cu_ReconImage);
        checkCudaErrors( hipMemcpy(dev_y+_nProjectionLength*nS, dev_v, _nProjectionSize, hipMemcpyDeviceToDevice ) );
    }

    ///////////////////////////////////////////////////////////////////////////////////////////////// Copy data from device to host ///////////////////////////////////////////////////////////////////////////////////////////////////
    checkCudaErrors(hipMemcpy(projection, dev_y, _nProjectionSize*_nSample, hipMemcpyDeviceToHost));

    //////////////////////////////////////////////////////////////////////////////////////////////////////// Cleanup Graphic card /////////////////////////////////////////////////////////////////////////////////////////////////////////
    checkCudaErrors(hipUnbindTexture(imgTexture2D));
    checkCudaErrors(hipFreeArray(cu_ReconImage));
    checkCudaErrors(hipFree(dev_y));
    checkCudaErrors(hipFree(d_pfDistIntersections));
    checkCudaErrors(hipFree(d_pfStart_1));
    checkCudaErrors(hipFree(d_pfStart_2));
    checkCudaErrors(hipFree(d_pfStep_1));
    checkCudaErrors(hipFree(d_pfStep_2));
    checkCudaErrors(hipFree(d_pnSteps));
    checkCudaErrors(hipFree(dev_x));
    checkCudaErrors(hipFree(dev_u));
    checkCudaErrors(hipFree(dev_v));
}


